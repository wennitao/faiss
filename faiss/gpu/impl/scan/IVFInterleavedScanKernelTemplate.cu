#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <faiss/MetricType.h>
#include <faiss/gpu/impl/scan/IVFInterleavedImpl.cuh>

namespace faiss {
namespace gpu {

template <>
void IVFINT_RUN<
        SUB_CODEC_TYPE,
        SUB_METRIC_TYPE,
        SUB_THREADS,
        SUB_NUM_WARP_Q,
        SUB_NUM_THREAD_Q>(
        SUB_CODEC_TYPE& codec,
        Tensor<float, 2, true>& queries,
        Tensor<idx_t, 2, true>& listIds,
        DeviceVector<void*>& listData,
        DeviceVector<void*>& listIndices,
        IndicesOptions indicesOptions,
        DeviceVector<idx_t>& listLengths,
        const int k,
        SUB_METRIC_TYPE metric,
        const bool useResidual,
        Tensor<float, 3, true>& residualBase,
        GpuScalarQuantizer* scalarQ,
        Tensor<float, 2, true>& outDistances,
        Tensor<idx_t, 2, true>& outIndices,
        GpuResources* res) {
    const auto nq = queries.getSize(0);
    const auto dim = queries.getSize(1);
    const auto nprobe = listIds.getSize(1);

    const auto stream = res->getDefaultStreamCurrentDevice();

    DeviceTensor<float, 3, true> distanceTemp(
            res,
            makeTempAlloc(AllocType::Other, stream),
            {queries.getSize(0), listIds.getSize(1), k});
    DeviceTensor<idx_t, 3, true> indicesTemp(
            res,
            makeTempAlloc(AllocType::Other, stream),
            {queries.getSize(0), listIds.getSize(1), k});

    const dim3 grid(nprobe, std::min(nq, (idx_t)getMaxGridCurrentDevice().y));
    // TEST ONLY!!!
    // const dim3 grid(nprobe, std::min(nq, (idx_t)getMaxGridCurrentDevice().y), 8);

    ivfInterleavedScan<
            SUB_CODEC_TYPE,
            SUB_METRIC_TYPE,
            SUB_THREADS,
            SUB_NUM_WARP_Q,
            SUB_NUM_THREAD_Q>
            <<<grid, SUB_THREADS, codec.getSmemSize(dim), stream>>>(
                    queries,
                    residualBase,
                    listIds,
                    listData.data(),
                    listLengths.data(),
                    codec,
                    metric,
                    k,
                    distanceTemp,
                    indicesTemp,
                    useResidual);

    runIVFInterleavedScan2(
            distanceTemp,
            indicesTemp,
            listIds,
            k,
            listIndices,
            indicesOptions,
            SUB_METRIC_TYPE::kDirection,
            outDistances,
            outIndices,
            stream);
}

template <>
void multiHeadIVFINT_RUN<
        SUB_CODEC_TYPE,
        SUB_METRIC_TYPE,
        SUB_THREADS,
        SUB_NUM_WARP_Q,
        SUB_NUM_THREAD_Q>(
        SUB_CODEC_TYPE& codec,
        const int nhead, 
        Tensor<float, 2, true>* queries,
        Tensor<idx_t, 2, true>* listIds,
        DeviceVector<void*>* listData,
        DeviceVector<void*>* listIndices,
        IndicesOptions indicesOptions,
        DeviceVector<idx_t>* listLengths,
        const int k,
        SUB_METRIC_TYPE metric,
        const bool useResidual,
        Tensor<float, 3, true>* residualBase,
        GpuScalarQuantizer* scalarQ,
        Tensor<float, 2, true>* outDistances,
        Tensor<idx_t, 2, true>* outIndices,
        GpuResources* res) {
    const auto nq = queries -> getSize(0);
    const auto dim = queries -> getSize(1);
    const auto nprobe = listIds -> getSize(1);

    const auto stream = res->getDefaultStreamCurrentDevice();

    // DeviceTensor<float, 3, true> distanceTemp(
    //         res,
    //         makeTempAlloc(AllocType::Other, stream),
    //         {queries -> getSize(0), listIds -> getSize(1), k});
    // DeviceTensor<idx_t, 3, true> indicesTemp(
    //         res,
    //         makeTempAlloc(AllocType::Other, stream),
    //         {queries -> getSize(0), listIds -> getSize(1), k});

    DeviceTensor<float, 3, true> distanceTemp[nhead] ;
    DeviceTensor<idx_t, 3, true> indicesTemp[nhead] ;

    for (int h = 0; h < nhead; h ++) {
        distanceTemp[h] = DeviceTensor<float, 3, true> (
                res, 
                makeTempAlloc(AllocType::Other, stream),
                {queries -> getSize(0), listIds -> getSize(1), k});
    }

    for (int h = 0; h < nhead; h ++) {
        indicesTemp[h] = DeviceTensor<idx_t, 3, true> (
                res, 
                makeTempAlloc(AllocType::Other, stream),
                {queries -> getSize(0), listIds -> getSize(1), k});
        // std::cerr << "debug: indicesTemp[" << h << "] size: " 
        //           << indicesTemp[h].getSize(0) << ", " 
        //           << indicesTemp[h].getSize(1) << ", " 
        //           << indicesTemp[h].getSize(2) << std::endl;
    }

    Tensor<float, 2, true>* devQueries ;
    Tensor<float, 3, true>* devResidualBase ;
    DeviceTensor<idx_t, 2, true>* devListIds ;

    DeviceTensor<float, 3, true>* devDistanceTemp ;
    DeviceTensor<idx_t, 3, true>* devIndicesTemp ;

    DeviceTensor<float, 2, true>* devOutDistances ;
    DeviceTensor<idx_t, 2, true>* devOutIndices ;

    hipMalloc((void**)&devQueries, nhead * sizeof(Tensor<float, 2, true>));
    hipMalloc((void**)&devResidualBase, nhead * sizeof(Tensor<float, 3, true>));
    hipMalloc((void**)&devListIds, nhead * sizeof(DeviceTensor<idx_t, 2, true>));

    hipMalloc((void**)&devDistanceTemp, nhead * sizeof(DeviceTensor<float, 3, true>));
    hipMalloc((void**)&devIndicesTemp, nhead * sizeof(DeviceTensor<idx_t, 3, true>));

    hipMalloc((void**)&devOutDistances, nhead * sizeof(DeviceTensor<float, 2, true>));
    hipMalloc((void**)&devOutIndices, nhead * sizeof(DeviceTensor<idx_t, 2, true>));

    hipMemcpy(devQueries, queries, nhead * sizeof(Tensor<float, 2, true>), hipMemcpyHostToDevice);
    hipMemcpy(devResidualBase, residualBase, nhead * sizeof(Tensor<float, 3, true>), hipMemcpyHostToDevice);
    hipMemcpy(devListIds, listIds, nhead * sizeof(DeviceTensor<idx_t, 2, true>), hipMemcpyHostToDevice);

    hipMemcpy(devDistanceTemp, distanceTemp, nhead * sizeof(DeviceTensor<float, 3, true>), hipMemcpyHostToDevice);
    hipMemcpy(devIndicesTemp, indicesTemp, nhead * sizeof(DeviceTensor<idx_t, 3, true>), hipMemcpyHostToDevice);

    hipMemcpy(devOutDistances, outDistances, nhead * sizeof(DeviceTensor<float, 2, true>), hipMemcpyHostToDevice);
    hipMemcpy(devOutIndices, outIndices, nhead * sizeof(DeviceTensor<idx_t, 2, true>), hipMemcpyHostToDevice);

    void*** listData_ptr = new void**[nhead];
    idx_t** listLengths_ptr = new idx_t*[nhead];
    for (int h = 0; h < nhead; h ++) {
        listData_ptr[h] = listData[h].data();
        listLengths_ptr[h] = listLengths[h].data();
    }

    void*** devListData; 
    idx_t** devListLengths;

    hipMalloc((void**)&devListData, nhead * sizeof(void**));
    hipMalloc((void**)&devListLengths, nhead * sizeof(idx_t*));
    hipMemcpy(devListData, listData_ptr, nhead * sizeof(void**), hipMemcpyHostToDevice);
    hipMemcpy(devListLengths, listLengths_ptr, nhead * sizeof(idx_t*), hipMemcpyHostToDevice);

    const dim3 grid(nprobe, std::min(nq, (idx_t)getMaxGridCurrentDevice().y), nhead);
    // const dim3 grid(nprobe, std::min(nq, (idx_t)getMaxGridCurrentDevice().y));

    multiHeadIvfInterleavedScan<
            SUB_CODEC_TYPE,
            SUB_METRIC_TYPE,
            SUB_THREADS,
            SUB_NUM_WARP_Q,
            SUB_NUM_THREAD_Q>
            <<<grid, SUB_THREADS, codec.getSmemSize(dim), stream>>>(
                    devQueries,
                    devResidualBase,
                    devListIds,
                    devListData,
                    devListLengths,
                    codec,
                    metric,
                    k,
                    devDistanceTemp,
                    devIndicesTemp,
                    useResidual);

//     std::cerr << "debug: multiHeadIvfInterleavedScan kernel launched" << std::endl;
//     for (int h = 0; h < nhead; h ++) {
//         std::cerr << "head " << h << std::endl ;
//         auto distanceTemp_vector = distanceTemp[h].copyToVector(stream);
//         auto indicesTemp_vector = indicesTemp[h].copyToVector(stream);
//         for (int i = 0; i < distanceTemp_vector.size(); i++) {
//             std::cerr << distanceTemp_vector[i] << " ";
//         }
//         std::cerr << std::endl;
//         for (int i = 0; i < indicesTemp_vector.size(); i++) {
//             std::cerr << indicesTemp_vector[i] << " ";
//         }
//         std::cerr << std::endl;
//     }

    runMultiHeadIVFInterleavedScan2(
            nhead, 
            nq, 
            devDistanceTemp,
            devIndicesTemp,
            devListIds,
            k,
            listIndices,
            indicesOptions,
            SUB_METRIC_TYPE::kDirection,
            devOutDistances,
            devOutIndices,
            stream);

    hipMemcpy(outDistances, devOutDistances, nhead * sizeof(DeviceTensor<float, 2, true>), hipMemcpyDeviceToHost);
    hipMemcpy(outIndices, devOutIndices, nhead * sizeof(DeviceTensor<idx_t, 2, true>), hipMemcpyDeviceToHost);
    
    hipFree(devQueries);
    hipFree(devResidualBase);
    hipFree(devListIds);
    hipFree(devDistanceTemp);
    hipFree(devIndicesTemp);
    hipFree(devOutDistances);
    hipFree(devOutIndices);
    hipFree(devListData);
    hipFree(devListLengths);

    delete[] listData_ptr;
    delete[] listLengths_ptr;
}

} // namespace gpu
} // namespace faiss
